#include "hip/hip_runtime.h"
/*************************************************************************************
LIBRARY: Connected-Component Labeling (CCL)
FILE:    main.cu
DATE:    2/11/2014
UPDATED: 

Contains the interface of CCL CPU alg. Finish your own gpu CCL alg in this file.
**************************************************************************************/

/**********************************************************************************************
***********************************************************************************************
#cat: gpuLabelImage - CCL GPU alg                       

Input:
w           - width of the image in pixels ÊäÈëÍ¼ÏñµÄÏñËØ¿í¶È X
h           - height of the image in pixels ÊäÈëÍ¼ÏñµÄÏñËØ¸ß¶È Y
ws          - pitch of the source image in bytes
wd          - pitch of the destination image in bytes 
img         - source image ÊäÈëÍ¼Ïñ
byF         - foreground mark (always 1 in this driver) Í¼ÏñµÄ±êÊ¶·û

Output:
numLabels   - The number of Labels (targets) in the image ÎïÌåµÄ¸öÊý
imgOut      - destination image Êä³öµÄÍ¼Ïñ

Return Codes:
reserved
**********************************************************************************************/
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"

#include <stdio.h>
#include <math.h>
#include <algorithm>
#include <ctime>
#include <cstdlib>

#include "ccl_gpu.cuh"

#define CUDA_CALL(x) {const hipError_t a = (x);if (a != hipSuccess) { printf("\nCuda error: %s (err_num = %d)\n",hipGetErrorString(a),a);hipDeviceReset();}}
#define MAX_HEIGHT 1024
#define MAX_WIDTH 1024
#define MAX_VERTEX MAX_WIDTH * MAX_HEIGHT

#define INT_PTR(x) (*((int*)(&(x))))




__global__ void get_vertex() {
	

}

__global__ void init_label() {

}

__global__ void CCL() {

}

__global__ void get_label_num() {

}

__global__ void set_image() {

}

int gpuLabelImage(int w, int h, int ws, int wd, unsigned char *img, int *imgOut, unsigned char byF,int *numLabels)
{
	
	bool flag = 0;
	int edge_num = 0;
	const bool false_flag = 0;

	bool *d_flag;
	int *d_edge_num;
	int *d_lable_num;
	int *d_label;
	int *d_vertex;

	hipMalloc(&d_flag, sizeof(bool));
	hipMalloc(&d_edge_num, sizeof(int));
	hipMalloc(&d_lable_num, sizeof(int));
	hipMalloc(&d_label, sizeof(int)*MAX_VERTEX);
	hipMalloc(&d_vertex, sizeof(int)*MAX_VERTEX*9);

	hipMemset(&d_vertex, 0, sizeof(int)*MAX_VERTEX*9);
	
	get_vertex<<<>>>();
	init_label<<<>>();

	do {
		CCL<<<>>();

	} while(flag == 1);

	get_label_num<<<>>();

	set_image<<<>>>();


	return 0;

}
 