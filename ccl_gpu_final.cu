/*************************************************************************************
LIBRARY: Connected-Component Labeling (CCL)
FILE:    main.cu
DATE:    2/11/2014
UPDATED: 

Contains the interface of CCL CPU alg. Finish your own gpu CCL alg in this file.
**************************************************************************************/

/**********************************************************************************************
***********************************************************************************************
#cat: gpuLabelImage - CCL GPU alg                       

Input:
w           - width of the image in pixels ÊäÈëÍ¼ÏñµÄÏñËØ¿í¶È X
h           - height of the image in pixels ÊäÈëÍ¼ÏñµÄÏñËØ¸ß¶È Y
ws          - pitch of the source image in bytes
wd          - pitch of the destination image in bytes 
img         - source image ÊäÈëÍ¼Ïñ
byF         - foreground mark (always 1 in this driver) Í¼ÏñµÄ±êÊ¶·û

Output:
numLabels   - The number of Labels (targets) in the image ÎïÌåµÄ¸öÊý
imgOut      - destination image Êä³öµÄÍ¼Ïñ

Return Codes:
reserved
**********************************************************************************************/
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"

#include <stdio.h>
#include <math.h>
#include <algorithm>
#include <ctime>
#include <iostream>
#include <cstdlib>
#include <algorithm>

#include "ccl_gpu.cuh"

#define CUDA_CALL(x) {const hipError_t a = (x);if (a != hipSuccess) { printf("\nCuda error: %s (err_num = %d)\n",hipGetErrorString(a),a);hipDeviceReset();}}
#define MAX_HEIGHT 1024
#define MAX_WIDTH 1024
//#define MAX_VERTEX MAX_WIDTH * MAX_HEIGHT

#define INT_PTR(x) (*((int*)(&(x))))

__device__ int dx[8] = {-1,0,1,-1,1,-1,0,1};
__device__ int dy[8] = {-1,-1,-1,0,0,1,1,1};

__device__ int get_loc() {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	return gridDim.x * blockDim.x * idy +idx;
}

__device__ int get_x() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int get_y() {
	return blockIdx.y * blockDim.y + threadIdx.y;
}

__device__ bool check_bound(int x, int y, int w, int h) {
	if (x>0&&y>0&&x<w&&y<h) return 1;
		else return 0;
}


int gpuLabelImage(int w, int h, int ws, int wd, unsigned char *img, int *imgOut, unsigned char byF,int *numLabels)
{


	return 0;
	
	
}
