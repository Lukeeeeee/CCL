


#include "hip/hip_runtime.h"


#include "device_atomic_functions.h"


__device__ int get_loc() {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	return gridDim.x * blockDim.x * idy +idx;
} 

__device__ int get_x() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int get_y() {
	return blockIdx.y * blockDim.y + threadIdx.y;
}

__device__ bool check_bound(int x, int y, int w, int h) {
	if (x>0&&y>0&&x<w&&y<h) return 1;
		else return 0;
}

__device__ bool check_connect(int loc1, int loc2, unsigned char *img, unsigned char byF) {
	if(img[loc1]==byF&&img[loc2]==byF) return 1;
	return 0;
}